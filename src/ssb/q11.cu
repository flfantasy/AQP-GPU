#include "hip/hip_runtime.h"
// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <iostream>
#include <stdio.h>
#include <hiprand.h>

#include <hip/hip_runtime.h>
#include <cub/util_allocator.cuh>
#include "cub/test/test_util.h"

#include "crystal/crystal.cuh"

#include "gpu_utils.h"
#include "ssb_utils.h"

using namespace std;

/**
 * Globals, constants and typedefs
 */
bool                    g_verbose = false;  // Whether to display input/output to console
hipcub::CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory, 用于给变量分配设备内存

template<int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void QueryKernel(int* lo_orderdate, int* lo_discount, int* lo_quantity, int* lo_extendedprice,
    int lo_num_entries, unsigned long long* revenue) {
  // Load a segment of consecutive items that are blocked across threads
  int items[ITEMS_PER_THREAD];
  int selection_flags[ITEMS_PER_THREAD];
  int items2[ITEMS_PER_THREAD];

  long long sum = 0;

  // 当前tile在整个数组中的offset
  int tile_offset = blockIdx.x * TILE_SIZE;
  // tile的数量
  int num_tiles = (lo_num_entries + TILE_SIZE - 1) / TILE_SIZE;
  // 当前tile内有多少items
  int num_tile_items = TILE_SIZE;
  if (blockIdx.x == num_tiles - 1) {
    num_tile_items = lo_num_entries - tile_offset;
  }

  // lo_orderdate >= 19930101 and lo_orderdate < 19940101
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_orderdate + tile_offset, items, num_tile_items);
  BlockPredGT<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 19930000, selection_flags, num_tile_items);
  BlockPredAndLT<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 19940000, selection_flags, num_tile_items);

  // lo_quantity<25
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_quantity + tile_offset, items, num_tile_items);
  BlockPredAndLT<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 25, selection_flags, num_tile_items);

  // lo_discount>=1 and lo_discount<=3
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_discount + tile_offset, items, num_tile_items);
  BlockPredAndGTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 1, selection_flags, num_tile_items);
  BlockPredAndLTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 3, selection_flags, num_tile_items);

  // lo_extendedprice
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_extendedprice + tile_offset, items2, num_tile_items);

  // 计算一个tile（128 * 4）的sum
  #pragma unroll
  for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM)
  {
    if ((threadIdx.x + (BLOCK_THREADS * ITEM) < num_tile_items))
      if (selection_flags[ITEM])
        sum += items[ITEM] * items2[ITEM];
  }

  __syncthreads();

  // 共享变量buffer存储中间结果
  static __shared__ long long buffer[32];
  
  unsigned long long aggregate = BlockSum<long long, BLOCK_THREADS, ITEMS_PER_THREAD>(sum, (long long*)buffer);
  __syncthreads();

  if (threadIdx.x == 0) {
    atomicAdd(revenue, aggregate);
  }
}

// 此函数主要是记录核函数的时间，传入四列数组（的显存地址）
float runQuery(int* lo_orderdate, int* lo_discount, int* lo_quantity, int* lo_extendedprice, 
    int lo_num_entries, hipcub::CachingDeviceAllocator&  g_allocator) {
  // 注册start、stop事件
  SETUP_TIMING();

  float time_query;
  chrono::high_resolution_clock::time_point st, finish;
  st = chrono::high_resolution_clock::now();

  hipEventRecord(start, 0);

  unsigned long long* d_sum = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_sum, sizeof(long long)));

  hipMemset(d_sum, 0, sizeof(long long));

  // Run
  int tile_items = 128*4;
  int num_blocks = (lo_num_entries + tile_items - 1)/tile_items;
  QueryKernel<128,4><<<num_blocks, 128>>>(lo_orderdate, 
          lo_discount, lo_quantity, lo_extendedprice, lo_num_entries, d_sum);

  // 计算时间
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time_query, start,stop);

  unsigned long long revenue;
  CubDebugExit(hipMemcpy(&revenue, d_sum, sizeof(long long), hipMemcpyDeviceToHost));

  finish = chrono::high_resolution_clock::now();
  std::chrono::duration<double> diff = finish - st;

  // diff = cpu数据传给gpu时间 + gpu执行时间
  cout << "Revenue: " << revenue << endl;
  cout << "Time Taken Total: " << diff.count() * 1000 << endl;

  CLEANUP(d_sum);
  // time_query = diff + 结果传回cpu时间
  return time_query;
}

/**
 * Main
 */
int main(int argc, char** argv)
{
  int num_trials          = 3;

  // Initialize command line
  CommandLineArgs args(argc, argv);
  args.GetCmdLineArgument("t", num_trials);

  // Print usage
  if (args.CheckCmdLineFlag("help"))
  {
      printf("%s "
          "[--t=<num trials>] "
          "[--v] "
          "\n", argv[0]);
      exit(0);
  }

  // Initialize device
  CubDebugExit(args.DeviceInit());

  // load column data to host memory
  int *h_lo_orderdate = loadColumn<int>("lo_orderdate", LO_LEN);
  int *h_lo_discount = loadColumn<int>("lo_discount", LO_LEN);
  int *h_lo_quantity = loadColumn<int>("lo_quantity", LO_LEN);
  int *h_lo_extendedprice = loadColumn<int>("lo_extendedprice", LO_LEN);
  int *h_d_datekey = loadColumn<int>("d_datekey", D_LEN);
  int *h_d_year = loadColumn<int>("d_year", D_LEN);

  cout << "** LOADED DATA **" << endl;
  cout << "LO_LEN " << LO_LEN << endl;

  // load column data to device memory
  int *d_lo_orderdate = loadToGPU<int>(h_lo_orderdate, LO_LEN, g_allocator);
  int *d_lo_discount = loadToGPU<int>(h_lo_discount, LO_LEN, g_allocator);
  int *d_lo_quantity = loadToGPU<int>(h_lo_quantity, LO_LEN, g_allocator);
  int *d_lo_extendedprice = loadToGPU<int>(h_lo_extendedprice, LO_LEN, g_allocator);
  int *d_d_datekey = loadToGPU<int>(h_d_datekey, D_LEN, g_allocator);
  int *d_d_year = loadToGPU<int>(h_d_year, D_LEN, g_allocator);

  cout << "** LOADED DATA TO GPU **" << endl;

  for (int t = 0; t < num_trials; t++) {
    // 查询执行时间
    float time_query;
    time_query = runQuery(d_lo_orderdate, d_lo_discount, d_lo_quantity, d_lo_extendedprice, LO_LEN, g_allocator);
    cout<< "{"
        << "\"query\":11" 
        << ",\"time_query\":" << time_query
        << "}" << endl;
  }

  return 0;
}

