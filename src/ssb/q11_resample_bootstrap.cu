#include "hip/hip_runtime.h"
// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR
#define TIMES 128
#define THREADS_PER_BLOCK 32
#define NUM_SM 72
#define WARPS_PER_SM 32
#define THREADS_PER_WARP 32
#define MAX_CONCURRENT_THREADS (NUM_SM * WARPS_PER_SM * THREADS_PER_WARP) 

#include <iostream>
#include <fstream>
#include <string>
#include <algorithm>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <cub/util_allocator.cuh>
#include "cub/test/test_util.h"

#include "crystal/crystal.cuh"

#include "gpu_utils.h"
#include "ssb_utils.h"

using namespace std;

// Caching allocator for device memory, 用于给变量分配设备内存
hipcub::CachingDeviceAllocator  g_allocator(true);  

static __device__ __inline__ uint32_t __mysmid(){
  uint32_t smid;
  asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));
  return smid;
}

static __device__ __inline__ uint32_t __mywarpid(){
  uint32_t warpid;
  asm volatile("mov.u32 %0, %%warpid;" : "=r"(warpid));
  return warpid;
}

static __device__ __inline__ uint32_t __mylaneid(){    
  uint32_t laneid;    
  asm volatile("mov.u32 %0, %%laneid;" : "=r"(laneid));    
  return laneid;
  }

__global__ void  curandGenKernel(hiprandState *curand_states,long clock_for_rand) {
  for(int i = 0; i < MAX_CONCURRENT_THREADS; i++) {
    hiprand_init(clock_for_rand + i, 0, 0, curand_states + i);
  }
}

// 进行一次BS试验的采样部分
__global__ void create_BS_sample(
    int* bs_lo_orderdate, int* bs_lo_discount, int* bs_lo_quantity, int* bs_lo_extendedprice,
    int* d_lo_orderdate, int* d_lo_discount, int* d_lo_quantity, int* d_lo_extendedprice,
    hiprandState *curand_states) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < LO_LEN) {
    int smid = __mysmid();
    int warpid = __mywarpid();
    int laneid = __mylaneid();
    unsigned int rand = hiprand(curand_states + (smid * 1024 + warpid * 32 + laneid));
    int loidx = rand % LO_LEN;
    bs_lo_orderdate[x] = d_lo_orderdate[loidx];
    bs_lo_discount[x] = d_lo_discount[loidx];
    bs_lo_quantity[x] = d_lo_quantity[loidx];
    bs_lo_extendedprice[x] = d_lo_extendedprice[loidx];    
  }
}

void run(int* h_lo_orderdate, int* h_lo_discount, int* h_lo_quantity, int* h_lo_extendedprice, 
  hipcub::CachingDeviceAllocator&  g_allocator) {
  // load column data to device memory
  int *d_lo_orderdate = loadToGPU<int>(h_lo_orderdate, LO_LEN, g_allocator);
  int *d_lo_discount = loadToGPU<int>(h_lo_discount, LO_LEN, g_allocator);
  int *d_lo_quantity = loadToGPU<int>(h_lo_quantity, LO_LEN, g_allocator);
  int *d_lo_extendedprice = loadToGPU<int>(h_lo_extendedprice, LO_LEN, g_allocator);

  // BS样本
  int* bs_lo_orderdate = NULL;
  int* bs_lo_discount = NULL;
  int* bs_lo_quantity = NULL;
  int* bs_lo_extendedprice = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&bs_lo_orderdate, LO_LEN * sizeof(int)));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&bs_lo_discount, LO_LEN * sizeof(int)));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&bs_lo_quantity, LO_LEN * sizeof(int)));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&bs_lo_extendedprice, LO_LEN * sizeof(int)));
  hipMemset(bs_lo_orderdate, 0, LO_LEN * sizeof(int));
  hipMemset(bs_lo_discount, 0, LO_LEN * sizeof(int));
  hipMemset(bs_lo_quantity, 0, LO_LEN * sizeof(int));
  hipMemset(bs_lo_extendedprice, 0, LO_LEN * sizeof(int));
  int* h_bs_lo_orderdate = new int[LO_LEN];
  int* h_bs_lo_discount = new int[LO_LEN];
  int* h_bs_lo_quantity = new int[LO_LEN];
  int* h_bs_lo_extendedprice = new int[LO_LEN];

  // cuda注册start、stop事件，用于计时
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float time1 = 0.0f;
  float unit_time = 0.0f;
  hipEventRecord(start, 0);
  // 随机数生成器初始化，为了防止冲突需要设置多个。titan的并行度是72个sm * 32个warp/sm * 32个thread/warp
  hiprandState *curand_state;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&curand_state, MAX_CONCURRENT_THREADS * sizeof(hiprandState)));
  long clock_for_rand = clock();  //程序运行时钟数
  curandGenKernel<<<1, 1>>>(curand_state, clock_for_rand);    //main road
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&unit_time, start, stop);
  time1 += unit_time;

  cout << "TIMES: " << TIMES << "[";
  for (int i = 0; i < TIMES; i++) {
    hipEventRecord(start, 0);
    {
      int num_blocks = (LO_LEN - 1) / THREADS_PER_BLOCK + 1;
      create_BS_sample<<<num_blocks, THREADS_PER_BLOCK>>>(bs_lo_orderdate, bs_lo_discount, bs_lo_quantity, bs_lo_extendedprice, d_lo_orderdate, d_lo_discount, d_lo_quantity, d_lo_extendedprice, curand_state);
      CubDebugExit(hipMemcpy(h_bs_lo_orderdate, bs_lo_orderdate, LO_LEN * sizeof(int), hipMemcpyDeviceToHost));
      CubDebugExit(hipMemcpy(h_bs_lo_discount, bs_lo_discount, LO_LEN * sizeof(int), hipMemcpyDeviceToHost));
      CubDebugExit(hipMemcpy(h_bs_lo_quantity, bs_lo_quantity, LO_LEN * sizeof(int), hipMemcpyDeviceToHost));
      CubDebugExit(hipMemcpy(h_bs_lo_extendedprice, bs_lo_extendedprice, LO_LEN * sizeof(int), hipMemcpyDeviceToHost));
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&unit_time, start, stop);
    time1 += unit_time;
    if (i % 10 == 0) cout << i;
    cout << "=" << flush;
  }
  cout << "]" << endl;
  cout << "Time Taken(resample): " << time1 << "ms" << endl;

  CLEANUP(curand_state);
  CLEANUP(bs_lo_orderdate);
  CLEANUP(bs_lo_discount);
  CLEANUP(bs_lo_quantity);
  CLEANUP(bs_lo_extendedprice);
}
/**
 * Main
 */
int main(int argc, char** argv){
  // Initialize command line
  CommandLineArgs args(argc, argv);

  // Initialize device
  CubDebugExit(args.DeviceInit());

  // load column data to host memory
  int *h_lo_orderdate = loadColumn<int>("lo_orderdate", LO_LEN);
  int *h_lo_discount = loadColumn<int>("lo_discount", LO_LEN);
  int *h_lo_quantity = loadColumn<int>("lo_quantity", LO_LEN);
  int *h_lo_extendedprice = loadColumn<int>("lo_extendedprice", LO_LEN);

  // 注册st、finish时间点，c++的计时工具
	chrono::high_resolution_clock::time_point st, finish;
  st = chrono::high_resolution_clock::now();
  run(h_lo_orderdate, h_lo_discount, h_lo_quantity, h_lo_extendedprice, g_allocator);
  finish = chrono::high_resolution_clock::now();
  chrono::duration<double> diff = finish - st;
  cout << "total time: " << diff.count() * 1000 << "ms" << endl;
  return 0;
}