#include "hip/hip_runtime.h"
// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR
#define TIMES 128
#define THREADS_PER_BLOCK 32
#define NUM_SM 72
#define WARPS_PER_SM 32
#define THREADS_PER_WARP 32
#define MAX_CONCURRENT_THREADS (NUM_SM * WARPS_PER_SM * THREADS_PER_WARP) 

#include <iostream>
#include <fstream>
#include <string>
#include <algorithm>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <cub/util_allocator.cuh>
#include "cub/test/test_util.h"

#include "crystal/crystal.cuh"

#include "gpu_utils.h"
#include "ssb_utils.h"

using namespace std;

// Caching allocator for device memory, 用于给变量分配设备内存
hipcub::CachingDeviceAllocator  g_allocator(true);  

static __device__ __inline__ uint32_t __mysmid(){
  uint32_t smid;
  asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));
  return smid;
}

static __device__ __inline__ uint32_t __mywarpid(){
  uint32_t warpid;
  asm volatile("mov.u32 %0, %%warpid;" : "=r"(warpid));
  return warpid;
}

static __device__ __inline__ uint32_t __mylaneid(){    
  uint32_t laneid;    
  asm volatile("mov.u32 %0, %%laneid;" : "=r"(laneid));    
  return laneid;
  }

__global__ void  curandGenKernel(hiprandState *curand_states,long clock_for_rand) {
  for(int i = 0; i < MAX_CONCURRENT_THREADS; i++) {
    hiprand_init(clock_for_rand + i, 0, 0, curand_states + i);
  }
}

// 进行一次BS试验的采样部分
__global__ void create_BS_sample(
    int* lo_linecount, hiprandState *curand_states) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < LO_LEN) {
    int smid = __mysmid();
    int warpid = __mywarpid();
    int laneid = __mylaneid();
    unsigned int rand = hiprand(curand_states + (smid * 1024 + warpid * 32 + laneid));
    int loidx = rand % LO_LEN;
    atomicAdd(&lo_linecount[loidx], 1);
  }
}

__global__ void memset_one(int* lo_linecount) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < LO_LEN) {
    lo_linecount[x] = 1;
  }
}

// 进行一次BS试验的查询部分，使用crystal
template<int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void queryKernel(int* lo_orderdate, int* lo_discount, int* lo_quantity, int* lo_extendedprice, int* lo_linecount,
    unsigned long long* res) {
  // items1表示某一列中由这个thread处理的几行
  // items2表示另一列中由这个thread处理的几行
  // selection_flags是一个bitmap，表示是否通过过滤
  int items1[ITEMS_PER_THREAD];
  int items2[ITEMS_PER_THREAD];
  int selection_flags[ITEMS_PER_THREAD];
  int linecount[ITEMS_PER_THREAD];

  long long sum = 0;

  // 当前tile在整个数组中的offset
  int tile_offset = blockIdx.x * TILE_SIZE;
  // tile的数量，LO_LEN/TILE_SIZE 向上取整
  int num_tiles = (LO_LEN + TILE_SIZE - 1) / TILE_SIZE;
  // 当前tile内有多少items
  int num_tile_items = TILE_SIZE;
  if (blockIdx.x == num_tiles - 1) {
    num_tile_items = LO_LEN - tile_offset;
  }

  // lo_orderdate >= 19930101 and lo_orderdate < 19940101
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_orderdate + tile_offset, items1, num_tile_items);
  BlockPredGT<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items1, 19930000, selection_flags, num_tile_items);
  BlockPredAndLT<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items1, 19940000, selection_flags, num_tile_items);

  // lo_quantity<25
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_quantity + tile_offset, items1, num_tile_items);
  BlockPredAndLT<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items1, 25, selection_flags, num_tile_items);

  // lo_discount>=1 and lo_discount<=3
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_discount + tile_offset, items1, num_tile_items);
  BlockPredAndGTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items1, 1, selection_flags, num_tile_items);
  BlockPredAndLTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items1, 3, selection_flags, num_tile_items);

  // lo_extendedprice
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_extendedprice + tile_offset, items2, num_tile_items);
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_linecount + tile_offset, linecount, num_tile_items);

  // 计算一个thread的sum
  #pragma unroll
  for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM)
  {
    if ((threadIdx.x + (BLOCK_THREADS * ITEM) < num_tile_items))
      if (selection_flags[ITEM])
        sum += items1[ITEM] * items2[ITEM] * linecount[ITEM];
  }
  __syncthreads();

  // buffer用于存储每个warp的sum，最多支持32个warp
  static __shared__ long long buffer[32];
  
  // 计算整个block的sum
  unsigned long long aggregate = BlockSum<long long, BLOCK_THREADS, ITEMS_PER_THREAD>(sum, (long long*)buffer);
  __syncthreads();

  if (threadIdx.x == 0) {
    atomicAdd(res, aggregate);
  }
}

void run(int* h_lo_orderdate, int* h_lo_discount, int* h_lo_quantity, int* h_lo_extendedprice, 
  hipcub::CachingDeviceAllocator&  g_allocator) {
  // load column data to device memory
  int *d_lo_orderdate = loadToGPU<int>(h_lo_orderdate, LO_LEN, g_allocator);
  int *d_lo_discount = loadToGPU<int>(h_lo_discount, LO_LEN, g_allocator);
  int *d_lo_quantity = loadToGPU<int>(h_lo_quantity, LO_LEN, g_allocator);
  int *d_lo_extendedprice = loadToGPU<int>(h_lo_extendedprice, LO_LEN, g_allocator);

  // 原表计数和BS采样计数
  int* lo_linecount = NULL;
  int* lo_linecount_origin = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&lo_linecount, LO_LEN * sizeof(int)));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&lo_linecount_origin, LO_LEN * sizeof(int)));
  hipMemset(lo_linecount, 0, LO_LEN * sizeof(int));
  hipMemset(lo_linecount_origin, 0, LO_LEN * sizeof(int));

  // 记录TIMES次bootstarp的sum
  unsigned long long* d_res = NULL;
  unsigned long long* d_bs_res = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_res, sizeof(unsigned long long)));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_bs_res, TIMES * sizeof(unsigned long long)));
  hipMemset(d_bs_res, 0, TIMES * sizeof(unsigned long long));
  unsigned long long h_res;
  unsigned long long h_bs_res[TIMES];

  // cuda注册start、stop事件，用于计时
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float time1 = 0.0f;
  float time2 = 0.0f;
  float unit_time = 0.0f;
  hipEventRecord(start, 0);
  // 随机数生成器初始化，为了防止冲突需要设置多个。titan的并行度是72个sm * 32个warp/sm * 32个thread/warp
  hiprandState *curand_state;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&curand_state, MAX_CONCURRENT_THREADS * sizeof(hiprandState)));
  long clock_for_rand = clock();  //程序运行时钟数
  curandGenKernel<<<1, 1>>>(curand_state, clock_for_rand);    //main road
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&unit_time, start, stop);
  time1 += unit_time;

  cout << "TIMES: " << TIMES << "[";
  for (int i = 0; i < TIMES; i++) {
    hipEventRecord(start, 0);
    {
      CubDebugExit(hipMemset(lo_linecount, 0, LO_LEN * sizeof(int)));
      int num_blocks = (LO_LEN - 1) / THREADS_PER_BLOCK + 1;
      create_BS_sample<<<num_blocks, THREADS_PER_BLOCK>>>(lo_linecount, curand_state);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&unit_time, start, stop);
    time1 += unit_time;
    if (i % 10 == 0) cout << i;
    cout << "=" << flush;
    hipEventRecord(start, 0);
    {
      int tile_items = 128*4;
      int num_blocks = (LO_LEN - 1) / tile_items + 1;
      queryKernel<128,4><<<num_blocks, 128>>>(d_lo_orderdate, d_lo_discount, d_lo_quantity, d_lo_extendedprice, lo_linecount, d_bs_res+i);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&unit_time, start, stop);
    time2 += unit_time;
  }
  cout << "]" << endl;
  cout << "Time Taken(resample): " << time1 << "ms" << endl;

  hipEventRecord(start, 0);
  CubDebugExit(hipMemcpy(&h_bs_res, d_bs_res, TIMES * sizeof(unsigned long long), hipMemcpyDeviceToHost));
  sort(h_bs_res, h_bs_res + TIMES);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&unit_time, start, stop);
  time2 += unit_time;

  int tile_items = 128*4;
  int num_blocks = (LO_LEN - 1) / THREADS_PER_BLOCK + 1;
  memset_one<<<num_blocks, THREADS_PER_BLOCK>>>(lo_linecount_origin);
  num_blocks = (LO_LEN - 1) / tile_items + 1;
  queryKernel<128,4><<<num_blocks, 128>>>(d_lo_orderdate, d_lo_discount, d_lo_quantity, d_lo_extendedprice, lo_linecount_origin, d_res);
  CubDebugExit(hipMemcpy(&h_res, d_res, sizeof(unsigned long long), hipMemcpyDeviceToHost));

  int idx1 = TIMES * 0.01;
  int idx2 = TIMES * 0.99;
  cout << h_res << " (" << (double)((long long)h_bs_res[idx1]-(long long)h_res)/h_res << ", " << (double)((long long)h_bs_res[idx2]-h_res)/h_res << ")" << endl; 
  cout << "Time Taken(run query): " << time2 << "ms" << endl;

  CLEANUP(curand_state);
}
/**
 * Main
 */
int main(int argc, char** argv){
  // Initialize command line
  CommandLineArgs args(argc, argv);

  // Initialize device
  CubDebugExit(args.DeviceInit());

  // load column data to host memory
  int *h_lo_orderdate = loadColumn<int>("lo_orderdate", LO_LEN);
  int *h_lo_discount = loadColumn<int>("lo_discount", LO_LEN);
  int *h_lo_quantity = loadColumn<int>("lo_quantity", LO_LEN);
  int *h_lo_extendedprice = loadColumn<int>("lo_extendedprice", LO_LEN);

  // 注册st、finish时间点，c++的计时工具
	chrono::high_resolution_clock::time_point st, finish;
  st = chrono::high_resolution_clock::now();
  run(h_lo_orderdate, h_lo_discount, h_lo_quantity, h_lo_extendedprice, g_allocator);
  finish = chrono::high_resolution_clock::now();
  chrono::duration<double> diff = finish - st;
  cout << "total time: " << diff.count() * 1000 << "ms" << endl;
  return 0;
}