#include "hip/hip_runtime.h"
/*调节threads_per_block和samples_per_thread数量
  比较速度快慢*/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR
#define TIMES 100
#define NUM_SM 72
#define WARPS_PER_SM 32
#define THREADS_PER_WARP 32
#define MOST_CONCURRENT_THREADS (NUM_SM * WARPS_PER_SM * THREADS_PER_WARP) 

#include <iostream>
#include <fstream>
#include <string>
#include <algorithm>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <cub/util_allocator.cuh>
#include "cub/test/test_util.h"

#include "gpu_utils.h"
#include "ssb_utils.h"

using namespace std;

hipcub::CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory, 用于给变量分配设备内存

static __device__ __inline__ uint32_t __mysmid(){
  uint32_t smid;
  asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));
  return smid;
}

static __device__ __inline__ uint32_t __mywarpid(){
  uint32_t warpid;
  asm volatile("mov.u32 %0, %%warpid;" : "=r"(warpid));
  return warpid;
}

static __device__ __inline__ uint32_t __mylaneid(){    
  uint32_t laneid;    
  asm volatile("mov.u32 %0, %%laneid;" : "=r"(laneid));    
  return laneid;
  }

__global__ void  curandGenKernel(hiprandState *curand_states,long clock_for_rand) {
  for(int i = 0; i < MOST_CONCURRENT_THREADS; i++) {
    hiprand_init(clock_for_rand + i, 0, 0, curand_states + i);
  }
}

// 向量化处理
__global__ void kernel(int* lo_revenue, int lo_num_entries, unsigned long long* sum,
    int entries_per_thread, hiprandState *curand_states) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int threads_per_bootstrap = (lo_num_entries - 1) / entries_per_thread + 1;
  if (x < threads_per_bootstrap * TIMES){
    int smid = __mysmid();
    int warpid = __mywarpid();
    int laneid = __mylaneid();
    int curandid = smid * 1024 + warpid * 32 + laneid;
    int sumTemp = 0;
    if (x / TIMES == threads_per_bootstrap - 1){
      int entries = (lo_num_entries - 1) % entries_per_thread + 1;
      for(int i = 0; i < entries; i++){
        sumTemp += lo_revenue[hiprand(curand_states + curandid) % lo_num_entries];
      }
    } else {
      for(int i = 0; i < entries_per_thread; i++){
        sumTemp += lo_revenue[hiprand(curand_states + curandid) % lo_num_entries];
      }
    }
    atomicAdd(&sum[x % TIMES], (unsigned long long)sumTemp); 
  }
}

// bootstrap的方法1：atomicadd很多，sum数组传回CPU再排序
void run(int* h_lo_revenue, int lo_num_entries, hipcub::CachingDeviceAllocator&  g_allocator,
  int threads_per_block, int entries_per_thread,
  unsigned long long* h_low_bound, unsigned long long* h_upper_bound) {

  float time_query;
  // cuda注册start、stop事件，用于计时
  hipEvent_t start, stop1, stop2, stop3, stop4, stop5;
{  hipEventCreate(&start);
  hipEventCreate(&stop1);
  hipEventCreate(&stop2);
  hipEventCreate(&stop3);
  hipEventCreate(&stop4);
  hipEventCreate(&stop5);
  hipEventRecord(start, 0);}

  // 数据从CPU传输到GPU
  int *d_lo_revenue = loadToGPU<int>(h_lo_revenue, lo_num_entries, g_allocator);
{  hipEventRecord(stop1, 0);
  hipEventSynchronize(stop1);
  hipEventElapsedTime(&time_query, start, stop1);
  cout << "H2D时间:" << time_query << "ms" << endl;}

  // 100次bootstarp的sum
  unsigned long long* d_sum = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_sum, 100 * sizeof(unsigned long long)));
  hipMemset(d_sum, 0, 100 * sizeof(unsigned long long));

  // 随机数生成器初始化，为了防止冲突需要设置多个。titan的并行度是72个sm * 32个warp/sm * 32个thread/warp
  hiprandState *curand_state;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&curand_state, MOST_CONCURRENT_THREADS * sizeof(hiprandState)));
  long clock_for_rand = clock();  //程序运行时钟数
  curandGenKernel<<<1, 1>>>(curand_state, clock_for_rand);    //main road
{  hipEventRecord(stop2, 0);
  hipEventSynchronize(stop2);
  hipEventElapsedTime(&time_query, stop1, stop2);
  cout << "随机数生成器初始化:" << time_query << "ms" << endl;}

  int num_blocks = (((lo_num_entries - 1) / entries_per_thread + 1) * TIMES - 1) / threads_per_block + 1;
  kernel<<<num_blocks, threads_per_block>>>(d_lo_revenue, lo_num_entries, d_sum, entries_per_thread, curand_state);    //main road
{  hipEventRecord(stop3, 0);
  hipEventSynchronize(stop3);
  hipEventElapsedTime(&time_query, stop2, stop3);
  cout << "GPU采样时间:" << time_query << "ms" << endl;}

{  unsigned long long h_sum[100];
  CubDebugExit(hipMemcpy(&h_sum, d_sum, 100 * sizeof(unsigned long long), hipMemcpyDeviceToHost));
  sort(h_sum, h_sum + 100);
  cout << h_sum[1] << "," << h_sum[98] << endl;
  *h_low_bound = h_sum[1];
  *h_upper_bound = h_sum[98];
  hipEventRecord(stop4, 0);
  hipEventSynchronize(stop4);
  hipEventElapsedTime(&time_query, stop3, stop4);
  cout << "D2H并排序时间:" << time_query << "ms" << endl;
  CLEANUP(d_sum);
  CLEANUP(curand_state);
  CLEANUP(d_lo_revenue);}
}

/**
 * Main
 */
int main(int argc, char** argv){
  // Initialize command line
  CommandLineArgs args(argc, argv);

  // Initialize device
  CubDebugExit(args.DeviceInit());

  // load column data to host memory
  int *h_lo_revenue = loadColumn<int>("lo_revenue", LO_LEN);
  
  // 计算原始样本sum
  long long sum = 0;
  for (int i = 0; i < LO_LEN; i++) {
    sum += h_lo_revenue[i];
  }

  // 置信区间结果
  unsigned long long low_bound = 0;
  unsigned long long upper_bound = 0;
  
  // 枚举实验
  int x = 16;
  int y = 100;
  int threads_per_block[x];
  for(int i = 0; i < x; i++){
    threads_per_block[i] = (i + 1) * 2;
  }
  int samples_per_thread[y];
  for(int i = 0; i < y; i++){
    samples_per_thread[i] = i + 1;
  }
  int time[x][y];
  // 注册st、finish时间点，c++的计时工具
	chrono::high_resolution_clock::time_point st, finish;
  for(int i = 0; i < x; i++){
    for(int j = 0; j < y; j++){
      cout << threads_per_block[i] << " " << samples_per_thread[j] << endl;
      st = chrono::high_resolution_clock::now();
      run(h_lo_revenue, LO_LEN, g_allocator, threads_per_block[i], samples_per_thread[j], &low_bound, &upper_bound);  //main road
      finish = chrono::high_resolution_clock::now();
      chrono::duration<double> diff = finish - st;
      cout << "总时间: " << diff.count() * 1000 << "ms" << endl;
      time[i][j] = diff.count() * 1000;
    }
  }
  for(int i = 0; i < x; i++){
    for(int j = 0; j < y; j++){
      cout << time[i][j] << " ";
    }
    cout << endl;
  }
  cout << sum << "(" << (long)low_bound - sum << "," << upper_bound - sum << ")" << endl;
  cout << sum << "(" << (double)((long)low_bound - sum)/sum << "," << (double)(upper_bound - sum)/sum << ")" << endl;

	return 0;
} 