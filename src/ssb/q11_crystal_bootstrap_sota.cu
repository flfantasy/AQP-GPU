#include "hip/hip_runtime.h"
// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR
#define TIMES 100
#define THREADS_PER_BLOCK 32
#define NUM_SM 72
#define WARPS_PER_SM 32
#define THREADS_PER_WARP 32
#define MAX_CONCURRENT_THREADS (NUM_SM * WARPS_PER_SM * THREADS_PER_WARP) 

#include <iostream>
#include <fstream>
#include <string>
#include <algorithm>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <cub/util_allocator.cuh>
#include "cub/test/test_util.h"

#include "crystal/crystal.cuh"

#include "gpu_utils.h"
#include "ssb_utils.h"

using namespace std;

// Caching allocator for device memory, 用于给变量分配设备内存
hipcub::CachingDeviceAllocator  g_allocator(true);  

static __device__ __inline__ uint32_t __mysmid(){
  uint32_t smid;
  asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));
  return smid;
}

static __device__ __inline__ uint32_t __mywarpid(){
  uint32_t warpid;
  asm volatile("mov.u32 %0, %%warpid;" : "=r"(warpid));
  return warpid;
}

static __device__ __inline__ uint32_t __mylaneid(){    
  uint32_t laneid;    
  asm volatile("mov.u32 %0, %%laneid;" : "=r"(laneid));    
  return laneid;
  }

__global__ void  curandGenKernel(hiprandState *curand_states,long clock_for_rand) {
  for(int i = 0; i < MAX_CONCURRENT_THREADS; i++) {
    hiprand_init(clock_for_rand + i, 0, 0, curand_states + i);
  }
}

// 进行一次BS试验的采样部分
__global__ void create_BS_sample(
    int* bs_lo_orderdate, int* bs_lo_discount, int* bs_lo_quantity, int* bs_lo_extendedprice,
    int* d_lo_orderdate, int* d_lo_discount, int* d_lo_quantity, int* d_lo_extendedprice,
    hiprandState *curand_states) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < TIMES) {
    for (size_t i = 0; i < LO_LEN; i++)
    {
      int smid = __mysmid();
      int warpid = __mywarpid();
      int laneid = __mylaneid();
      unsigned int rand = hiprand(curand_states + (smid * 1024 + warpid * 32 + laneid));
      int loidx = rand % LO_LEN;
      int idx = x * LO_LEN + i;
      bs_lo_orderdate[idx] = d_lo_orderdate[loidx];
      bs_lo_discount[idx] = d_lo_discount[loidx];
      bs_lo_quantity[idx] = d_lo_quantity[loidx];
      bs_lo_extendedprice[idx] = d_lo_extendedprice[loidx];
    }
  }
}

// 进行一次BS试验的查询部分，使用crystal
template<int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void queryKernel(int* lo_orderdate, int* lo_discount, int* lo_quantity, int* lo_extendedprice,
    unsigned long long* res) {
  // items1表示某一列中由这个thread处理的几行
  // items2表示另一列中由这个thread处理的几行
  // selection_flags是一个bitmap，表示是否通过过滤
  int items1[ITEMS_PER_THREAD];
  int items2[ITEMS_PER_THREAD];
  int selection_flags[ITEMS_PER_THREAD];

  long long sum = 0;

  // 当前tile在整个数组中的offset
  int tile_offset = blockIdx.x * TILE_SIZE;
  // tile的数量，LO_LEN/TILE_SIZE 向上取整
  int num_tiles = (LO_LEN + TILE_SIZE - 1) / TILE_SIZE;
  // 当前tile内有多少items
  int num_tile_items = TILE_SIZE;
  if (blockIdx.x == num_tiles - 1) {
    num_tile_items = LO_LEN - tile_offset;
  }

  // lo_orderdate >= 19930101 and lo_orderdate < 19940101
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_orderdate + tile_offset, items1, num_tile_items);
  BlockPredGT<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items1, 19930000, selection_flags, num_tile_items);
  BlockPredAndLT<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items1, 19940000, selection_flags, num_tile_items);

  // lo_quantity<25
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_quantity + tile_offset, items1, num_tile_items);
  BlockPredAndLT<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items1, 25, selection_flags, num_tile_items);

  // lo_discount>=1 and lo_discount<=3
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_discount + tile_offset, items1, num_tile_items);
  BlockPredAndGTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items1, 1, selection_flags, num_tile_items);
  BlockPredAndLTE<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items1, 3, selection_flags, num_tile_items);

  // lo_extendedprice
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_extendedprice + tile_offset, items2, num_tile_items);

  // 计算一个thread的sum
  #pragma unroll
  for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM)
  {
    if ((threadIdx.x + (BLOCK_THREADS * ITEM) < num_tile_items))
      if (selection_flags[ITEM])
        sum += items1[ITEM] * items2[ITEM];
  }
  __syncthreads();

  // buffer用于存储每个warp的sum，最多支持32个warp
  static __shared__ long long buffer[32];
  
  // 计算整个block的sum
  unsigned long long aggregate = BlockSum<long long, BLOCK_THREADS, ITEMS_PER_THREAD>(sum, (long long*)buffer);
  __syncthreads();

  if (threadIdx.x == 0) {
    atomicAdd(res, aggregate);
  }
}

void run(int* h_lo_orderdate, int* h_lo_discount, int* h_lo_quantity, int* h_lo_extendedprice, 
  hipcub::CachingDeviceAllocator&  g_allocator) {
  // load column data to device memory
  int *d_lo_orderdate = loadToGPU<int>(h_lo_orderdate, LO_LEN, g_allocator);
  int *d_lo_discount = loadToGPU<int>(h_lo_discount, LO_LEN, g_allocator);
  int *d_lo_quantity = loadToGPU<int>(h_lo_quantity, LO_LEN, g_allocator);
  int *d_lo_extendedprice = loadToGPU<int>(h_lo_extendedprice, LO_LEN, g_allocator);

  // BS样本
  int* bs_lo_orderdate = NULL;
  int* bs_lo_discount = NULL;
  int* bs_lo_quantity = NULL;
  int* bs_lo_extendedprice = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&bs_lo_orderdate, TIMES * LO_LEN * sizeof(int)));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&bs_lo_discount, TIMES * LO_LEN * sizeof(int)));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&bs_lo_quantity, TIMES * LO_LEN * sizeof(int)));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&bs_lo_extendedprice, TIMES * LO_LEN * sizeof(int)));
  hipMemset(bs_lo_orderdate, 0, TIMES * LO_LEN * sizeof(int));
  hipMemset(bs_lo_discount, 0, TIMES * LO_LEN * sizeof(int));
  hipMemset(bs_lo_quantity, 0, TIMES * LO_LEN * sizeof(int));
  hipMemset(bs_lo_extendedprice, 0, TIMES * LO_LEN * sizeof(int));

  // 记录TIMES次bootstarp的sum
  unsigned long long* d_res = NULL;
  unsigned long long* d_bs_res = NULL;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_res, sizeof(unsigned long long)));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_bs_res, TIMES * sizeof(unsigned long long)));
  hipMemset(d_bs_res, 0, TIMES * sizeof(unsigned long long));
  unsigned long long h_res;
  unsigned long long h_bs_res[TIMES];

  // cuda注册start、stop事件，用于计时
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float time1 = 0.0f;
  float time2 = 0.0f;
  float unit_time = 0.0f;
  hipEventRecord(start, 0);
  // 随机数生成器初始化，为了防止冲突需要设置多个。titan的并行度是72个sm * 32个warp/sm * 32个thread/warp
  hiprandState *curand_state;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&curand_state, MAX_CONCURRENT_THREADS * sizeof(hiprandState)));
  long clock_for_rand = clock();  //程序运行时钟数
  curandGenKernel<<<1, 1>>>(curand_state, clock_for_rand);    //main road
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&unit_time, start, stop);
  time1 += unit_time;

  hipEventRecord(start, 0);
  {
    int num_blocks = (TIMES - 1) / THREADS_PER_BLOCK + 1;
    create_BS_sample<<<num_blocks, THREADS_PER_BLOCK>>>(bs_lo_orderdate, bs_lo_discount, bs_lo_quantity, bs_lo_extendedprice, d_lo_orderdate, d_lo_discount, d_lo_quantity, d_lo_extendedprice, curand_state);
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&unit_time, start, stop);
  time1 += unit_time;
  for (int i = 0; i < TIMES; i++) {
    hipEventRecord(start, 0);
    {
      int tile_items = 128*4;
      int num_blocks = (LO_LEN - 1) / tile_items + 1;
      int offset = i * LO_LEN;
      queryKernel<128,4><<<num_blocks, 128>>>(bs_lo_orderdate + offset, bs_lo_discount + offset, bs_lo_quantity + offset, bs_lo_extendedprice + offset, d_bs_res+i);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&unit_time, start, stop);
    time2 += unit_time;
  }
  cout << "]" << endl;
  cout << "Time Taken(resample): " << time1 << "ms" << endl;

  hipEventRecord(start, 0);
  CubDebugExit(hipMemcpy(&h_bs_res, d_bs_res, TIMES * sizeof(unsigned long long), hipMemcpyDeviceToHost));
  sort(h_bs_res, h_bs_res + TIMES);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&unit_time, start, stop);
  time2 += unit_time;

  int tile_items = 128*4;
  int num_blocks = (LO_LEN - 1) / tile_items + 1;
  queryKernel<128,4><<<num_blocks, 128>>>(d_lo_orderdate, d_lo_discount, d_lo_quantity, d_lo_extendedprice, d_res);
  CubDebugExit(hipMemcpy(&h_res, d_res, sizeof(unsigned long long), hipMemcpyDeviceToHost));

  int idx1 = TIMES * 0.01;
  int idx2 = TIMES * 0.99;
  cout << h_res << " (" << (double)((long long)h_bs_res[idx1]-(long long)h_res)/h_res << ", " << (double)((long long)h_bs_res[idx2]-h_res)/h_res << ")" << endl; 
  cout << "Time Taken(run query): " << time2 << "ms" << endl;

  CLEANUP(curand_state);
  CLEANUP(bs_lo_orderdate);
  CLEANUP(bs_lo_discount);
  CLEANUP(bs_lo_quantity);
  CLEANUP(bs_lo_extendedprice);
}
/**
 * Main
 */
int main(int argc, char** argv){
  // Initialize command line
  CommandLineArgs args(argc, argv);

  // Initialize device
  CubDebugExit(args.DeviceInit());

  // load column data to host memory
  int *h_lo_orderdate = loadColumn<int>("lo_orderdate", LO_LEN);
  int *h_lo_discount = loadColumn<int>("lo_discount", LO_LEN);
  int *h_lo_quantity = loadColumn<int>("lo_quantity", LO_LEN);
  int *h_lo_extendedprice = loadColumn<int>("lo_extendedprice", LO_LEN);

  // 注册st、finish时间点，c++的计时工具
	chrono::high_resolution_clock::time_point st, finish;
  st = chrono::high_resolution_clock::now();
  run(h_lo_orderdate, h_lo_discount, h_lo_quantity, h_lo_extendedprice, g_allocator);
  finish = chrono::high_resolution_clock::now();
  chrono::duration<double> diff = finish - st;
  cout << "total time: " << diff.count() * 1000 << "ms" << endl;
  return 0;
}