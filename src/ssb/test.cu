#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>
#include <iostream>
#include <stdlib.h>

#define blockSize 1024

#include <time.h>
#include <sys/time.h>
#define USECPSEC 1000000ULL

#define MAX_SM 72

unsigned long long dtime_usec(unsigned long long start){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}


__device__ unsigned long long count = 0;
__device__ unsigned int blk_ids[MAX_SM] = {0};

__global__ void rng_init(unsigned long long seed, hiprandState * states) {
  const size_t Idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed, Idx, 0, &states[Idx]);
}

__global__ void kernel(hiprandState * states, int length) {
  const size_t Idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = 0; i < length; i++){
    const float x = hiprand_uniform(&states[Idx]);
    const float y = hiprand_uniform(&states[Idx]);
    if (sqrtf(x*x+y*y)<1.0)
      atomicAdd(&count, 1ULL);}
}

static __device__ __inline__ int __mysmid(){
  int smid;
  asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));
  return smid;}

__device__ int get_my_resident_thread_id(int sm_blk_id){
  int my_sm = __mysmid();
  if (sm_blk_id != 0) {
    printf("smid: %d, block_id: %d\n", my_sm, sm_blk_id);
  }
  return my_sm * sm_blk_id + threadIdx.x;
}

__device__ int get_block_id(){
  int my_sm = __mysmid();
  int my_block_id = -1;
  bool done = false;
  int i = 0;
  while ((!done)&&(i<32)){
    unsigned int block_flag = 1<<i;
    if ((atomicOr(blk_ids+my_sm, block_flag)&block_flag) == 0){my_block_id = i; done = true;}
    i++;}
  return my_block_id;
}

__device__ void release_block_id(int block_id){
  unsigned int block_mask = ~(1<<block_id);
  int my_sm = __mysmid();
  atomicAnd(blk_ids+my_sm, block_mask);
}

__global__ void kernel2(hiprandState * states, int length) {

  __shared__ volatile int my_block_id;
  if (!threadIdx.x) my_block_id = get_block_id();
  __syncthreads();
  if (my_block_id != 0) {printf("%d\n",my_block_id);}
  const size_t Idx = get_my_resident_thread_id(my_block_id);
  for (int i = 0; i < length; i++){
    const float x = hiprand_uniform(&states[Idx]);
    const float y = hiprand_uniform(&states[Idx]);
    if (sqrtf(x*x+y*y)<1.0)
      atomicAdd(&count, 1ULL);}
  __syncthreads();
  if (!threadIdx.x) release_block_id(my_block_id);
  __syncthreads();
}



int main(int argc, char *argv[]) {
  int gridSize = 10;
  if (argc > 1) gridSize = atoi(argv[1]);
  hiprandState * states;
  assert(hipMalloc(&states, gridSize*gridSize*blockSize*sizeof(hiprandState)) == hipSuccess);
  unsigned long long hcount;
  //warm - up
  rng_init<<<gridSize*gridSize,blockSize>>>(1234ULL, states);
  assert(hipDeviceSynchronize() == hipSuccess);
  //method 1: 1 curand state per point
  std::cout << "Method 1 init blocks: " << gridSize*gridSize << std::endl;
  unsigned long long dtime = dtime_usec(0);
  rng_init<<<gridSize*gridSize,blockSize>>>(1234ULL, states);
  assert(hipDeviceSynchronize() == hipSuccess);
  unsigned long long initt = dtime_usec(dtime);
  kernel<<<gridSize*gridSize,blockSize>>>(states, 1);
  assert(hipDeviceSynchronize() == hipSuccess);
  dtime = dtime_usec(dtime) - initt;
  hipMemcpyFromSymbol(&hcount, HIP_SYMBOL(count), sizeof(unsigned long long));
  std::cout << "method 1 elapsed time: " << dtime/(float)USECPSEC << " init time: " << initt/(float)USECPSEC << " pi: " << 4.0f*hcount/(float)(gridSize*gridSize*blockSize) << std::endl;
  hcount = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(count), &hcount, sizeof(unsigned long long));
  //method 2: 1 curand state per gridSize points
  std::cout << "Method 2 init blocks: " << gridSize << std::endl;
  dtime = dtime_usec(0);
  rng_init<<<gridSize,blockSize>>>(1234ULL, states);
  assert(hipDeviceSynchronize() == hipSuccess);
  initt = dtime_usec(dtime);
  kernel<<<gridSize,blockSize>>>(states, gridSize);
  assert(hipDeviceSynchronize() == hipSuccess);
  dtime = dtime_usec(dtime) - initt;
  hipMemcpyFromSymbol(&hcount, HIP_SYMBOL(count), sizeof(unsigned long long));
  std::cout << "method 2 elapsed time: " << dtime/(float)USECPSEC << " init time: " << initt/(float)USECPSEC << " pi: " << 4.0f*hcount/(float)(gridSize*gridSize*blockSize) << std::endl;
  hcount = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(count), &hcount, sizeof(unsigned long long));
  //method 3: 1 curand state per resident thread
  // compute the maximum number of state entries needed
  int num_sms;
  hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
  int max_sm_threads;
  hipDeviceGetAttribute(&max_sm_threads, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
  int max_blocks = max_sm_threads/blockSize;
  int total_state = max_blocks*num_sms*blockSize;
  int rgridSize = (total_state + blockSize-1)/blockSize;
  std::cout << "Method 3 sms: " << num_sms << " init blocks: " << rgridSize << std::endl;
  // run test
  dtime = dtime_usec(0);
  rng_init<<<rgridSize,blockSize>>>(1234ULL, states);
  assert(hipDeviceSynchronize() == hipSuccess);
  initt = dtime_usec(dtime);
  kernel2<<<gridSize,blockSize>>>(states, gridSize);
  assert(hipDeviceSynchronize() == hipSuccess);
  dtime = dtime_usec(dtime) - initt;
  hipMemcpyFromSymbol(&hcount, HIP_SYMBOL(count), sizeof(unsigned long long));
  std::cout << "method 3 elapsed time: " << dtime/(float)USECPSEC << " init time: " << initt/(float)USECPSEC << " pi: " << 4.0f*hcount/(float)(gridSize*gridSize*blockSize) << std::endl;
  hcount = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(count), &hcount, sizeof(unsigned long long));
  return 0;
}