#include "hip/hip_runtime.h"
// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR
#define TIMES 128
#define THREADS_PER_BLOCK 32
#define NUM_SM 72
#define WARPS_PER_SM 32
#define THREADS_PER_WARP 32
#define MAX_CONCURRENT_THREADS (NUM_SM * WARPS_PER_SM * THREADS_PER_WARP) 

#include <iostream>
#include <fstream>
#include <string>
#include <algorithm>
#include <time.h>
#include <unordered_map>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <cub/util_allocator.cuh>
#include "cub/test/test_util.h"

#include "crystal/crystal.cuh"

#include "gpu_utils.h"
#include "ssb_utils.h"

using namespace std;

struct pair_hash
{
    template<class T1, class T2>
    size_t operator() (const pair<T1, T2>& p) const
    {
        auto h1 = hash<T1>{}(p.first);
        auto h2 = hash<T2>{}(p.second);
        return h1 ^ h2;
    }
};

// Caching allocator for device memory, 用于给变量分配设备内存
hipcub::CachingDeviceAllocator  g_allocator(true);  

static __device__ __inline__ uint32_t __mysmid(){
  uint32_t smid;
  asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));
  return smid;
}

static __device__ __inline__ uint32_t __mywarpid(){
  uint32_t warpid;
  asm volatile("mov.u32 %0, %%warpid;" : "=r"(warpid));
  return warpid;
}

static __device__ __inline__ uint32_t __mylaneid(){    
  uint32_t laneid;    
  asm volatile("mov.u32 %0, %%laneid;" : "=r"(laneid));    
  return laneid;
  }

__global__ void  curandGenKernel(hiprandState *curand_states,long clock_for_rand) {
  for(int i = 0; i < MAX_CONCURRENT_THREADS; i++) {
    hiprand_init(clock_for_rand + i, 0, 0, curand_states + i);
  }
}

template<int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void build_hashtable_c(int* filter_col, int *dim_key, int* dim_val, int *hash_table) {
  int items[ITEMS_PER_THREAD];
  int items2[ITEMS_PER_THREAD];
  int selection_flags[ITEMS_PER_THREAD];

  int tile_offset = blockIdx.x * TILE_SIZE;
  int num_tiles = (C_LEN + TILE_SIZE - 1) / TILE_SIZE;
  int num_tile_items = TILE_SIZE;

  if (blockIdx.x == num_tiles - 1) {
    num_tile_items = C_LEN - tile_offset;
  }

  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(filter_col + tile_offset, items, num_tile_items);
  BlockPredEQ<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 1, selection_flags, num_tile_items);

  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_key + tile_offset, items, num_tile_items);
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_val + tile_offset, items2, num_tile_items);
  BlockBuildSelectivePHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, items2, selection_flags, 
      hash_table, C_SLOT_LEN, num_tile_items);
}

template<int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void build_hashtable_s(int *filter_col, int *dim_key, int *hash_table) {
  int items[ITEMS_PER_THREAD];
  int selection_flags[ITEMS_PER_THREAD];

  int tile_offset = blockIdx.x * TILE_SIZE;
  int num_tiles = (S_LEN + TILE_SIZE - 1) / TILE_SIZE;
  int num_tile_items = TILE_SIZE;

  if (blockIdx.x == num_tiles - 1) {
    num_tile_items = S_LEN - tile_offset;
  }

  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(filter_col + tile_offset, items, num_tile_items);
  BlockPredEQ<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 1, selection_flags, num_tile_items);

  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_key + tile_offset, items, num_tile_items);
  BlockBuildSelectivePHT_1<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, selection_flags, 
      hash_table, S_SLOT_LEN, num_tile_items);
}

template<int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void build_hashtable_p(int *filter_col, int *dim_key, int *hash_table) {
  int items[ITEMS_PER_THREAD];
  int selection_flags[ITEMS_PER_THREAD];

  int tile_offset = blockIdx.x * TILE_SIZE;
  int num_tiles = (P_LEN + TILE_SIZE - 1) / TILE_SIZE;
  int num_tile_items = TILE_SIZE;

  if (blockIdx.x == num_tiles - 1) {
    num_tile_items = P_LEN - tile_offset;
  }

  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(filter_col + tile_offset, items, num_tile_items);
  BlockPredEQ<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 0, selection_flags, num_tile_items);
  BlockPredOrEQ<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, 1, selection_flags, num_tile_items);

  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_key + tile_offset, items, num_tile_items);
  BlockBuildSelectivePHT_1<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, selection_flags, 
      hash_table, P_SLOT_LEN, num_tile_items);
}

template<int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void build_hashtable_d(int *dim_key, int *dim_val, int *hash_table) {
  int items[ITEMS_PER_THREAD];
  int items2[ITEMS_PER_THREAD];
  int selection_flags[ITEMS_PER_THREAD];

  int tile_offset = blockIdx.x * TILE_SIZE;
  int num_tiles = (D_LEN + TILE_SIZE - 1) / TILE_SIZE;
  int num_tile_items = TILE_SIZE;

  if (blockIdx.x == num_tiles - 1) {
    num_tile_items = D_LEN - tile_offset;
  }

  InitFlags<BLOCK_THREADS, ITEMS_PER_THREAD>(selection_flags);
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_key + tile_offset, items, num_tile_items);
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(dim_val + tile_offset, items2, num_tile_items);
  BlockBuildSelectivePHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, items2, selection_flags,
      hash_table, D_SLOT_LEN, D_VAL_MIN, num_tile_items);
}

template<int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void probe(int* lo_orderdate, int* lo_custkey, int* lo_suppkey, int* lo_partkey, int* lo_revenue, int* lo_supplycost,
    int* ht_c, int* ht_s, int* ht_p, int* ht_d, int* res) {
  // Load a segment of consecutive items that are blocked across threads
  int items[ITEMS_PER_THREAD];
  int selection_flags[ITEMS_PER_THREAD];
  int c_nation[ITEMS_PER_THREAD];
  int d_year[ITEMS_PER_THREAD];
  int revenue[ITEMS_PER_THREAD];

  int tile_offset = blockIdx.x * TILE_SIZE;
  int num_tiles = (LO_LEN + TILE_SIZE - 1) / TILE_SIZE;
  int num_tile_items = TILE_SIZE;

  if (blockIdx.x == num_tiles - 1) {
    num_tile_items = LO_LEN - tile_offset;
  }

  InitFlags<BLOCK_THREADS, ITEMS_PER_THREAD>(selection_flags);

  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_suppkey + tile_offset, items, num_tile_items);
  BlockProbeAndPHT_1<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, selection_flags, ht_s, S_SLOT_LEN, num_tile_items);

  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_custkey + tile_offset, items, num_tile_items);
  BlockProbeAndPHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, c_nation, selection_flags,
      ht_c, C_SLOT_LEN, num_tile_items);

  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_partkey + tile_offset, items, num_tile_items);
  BlockProbeAndPHT_1<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, selection_flags, ht_p, P_SLOT_LEN, num_tile_items);

  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_orderdate + tile_offset, items, num_tile_items);
  BlockProbeAndPHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, d_year, selection_flags,
      ht_d, D_SLOT_LEN, D_VAL_MIN, num_tile_items);

  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_revenue + tile_offset, revenue, num_tile_items);
  BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD>(lo_supplycost + tile_offset, items, num_tile_items);

  #pragma unroll
  for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM) {
    if (threadIdx.x + (BLOCK_THREADS * ITEM) < num_tile_items) {
      if (selection_flags[ITEM]) {
        int hash = (c_nation[ITEM] * 7 +  (d_year[ITEM] - 1992)) % ((1998-1992+1) * 25);
        res[hash * 4] = d_year[ITEM];
        res[hash * 4 + 1] = c_nation[ITEM];
        atomicAdd(reinterpret_cast<unsigned long long*>(&res[hash * 4 + 2]), (long long)(revenue[ITEM] - items[ITEM]));
      }
    }
  }
}

template<int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void createBSSampleAndProbe(
    int* lo_orderdate, int* lo_custkey, int* lo_suppkey, int* lo_partkey, int* lo_revenue, int* lo_supplycost,
    int* ht_c, int* ht_s, int* ht_p, int* ht_d,
    int* res, hiprandState* curand_states) {
  // Load a tile striped across threads
  int items[ITEMS_PER_THREAD];
  int selection_flags[ITEMS_PER_THREAD];
  int c_nation[ITEMS_PER_THREAD];
  int d_year[ITEMS_PER_THREAD];
  int revenue[ITEMS_PER_THREAD];

  int tile_offset = blockIdx.x * TILE_SIZE;
  int num_tiles = (LO_LEN + TILE_SIZE - 1) / TILE_SIZE;
  int num_tile_items = TILE_SIZE;

  if (blockIdx.x == num_tiles - 1) {
    num_tile_items = LO_LEN - tile_offset;
  }

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < LO_LEN) {
    int smid = __mysmid();
    int warpid = __mywarpid();
    int laneid = __mylaneid();
    unsigned int rand = hiprand(curand_states + (smid * 1024 + warpid * 32 + laneid));
    int loidx = rand % LO_LEN; 
    InitFlags<BLOCK_THREADS, ITEMS_PER_THREAD>(selection_flags);
    items[0] = lo_suppkey[loidx];
    BlockProbeAndPHT_1<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, selection_flags, ht_s, S_SLOT_LEN, num_tile_items);
    items[0] = lo_custkey[loidx];
    BlockProbeAndPHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, c_nation, selection_flags,
        ht_c, C_SLOT_LEN, num_tile_items);
    items[0] = lo_partkey[loidx];
    BlockProbeAndPHT_1<int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, selection_flags, ht_p, P_SLOT_LEN, num_tile_items);
    items[0] = lo_orderdate[loidx];
    BlockProbeAndPHT_2<int, int, BLOCK_THREADS, ITEMS_PER_THREAD>(items, d_year, selection_flags,
        ht_d, D_SLOT_LEN, D_VAL_MIN, num_tile_items);
    revenue[0] = lo_revenue[loidx];
    items[0] = lo_supplycost[loidx];
    if (selection_flags[0]) {
      int hash = (c_nation[0] * 7 +  (d_year[0] - 1992)) % ((1998-1992+1) * 25);
      res[hash * 4] = d_year[0];
      res[hash * 4 + 1] = c_nation[0];
      atomicAdd(reinterpret_cast<unsigned long long*>(&res[hash * 4 + 2]), (long long)(revenue[0] - items[0]));
    }
  }
}

void createBSSampleAndRunQuery(
    int* lo_orderdate, int* lo_custkey, int* lo_suppkey, int* lo_partkey, int* lo_revenue, int* lo_supplycost,
    int* c_custkey, int* c_nation, int* c_region,
    int* s_suppkey, int* s_region,
    int* p_partkey, int* p_mfgr,
    int* d_datekey, int* d_year,
    unordered_map<pair<int, int>, vector<long long>, pair_hash>* bs_umap) {

  // 存储单次查询结果
  int* d_res;
  int res_size = ((1998-1992+1) * 25);
  int res_array_size = res_size * 4;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_res, res_array_size * sizeof(int)));
  CubDebugExit(hipMemset(d_res, 0, res_array_size * sizeof(int)));
  int* h_res = new int[res_array_size];

  // 三张hash表
  int *d_ht_c, *d_ht_s, *d_ht_p, *d_ht_d;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_ht_c, 2 * C_SLOT_LEN * sizeof(int)));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_ht_s, 2 * S_SLOT_LEN * sizeof(int)));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_ht_p, 2 * P_SLOT_LEN * sizeof(int)));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_ht_d, 2 * D_SLOT_LEN * sizeof(int)));
  CubDebugExit(hipMemset(d_ht_c, 0, 2 * C_SLOT_LEN * sizeof(int)));
  CubDebugExit(hipMemset(d_ht_s, 0, 2 * S_SLOT_LEN * sizeof(int)));
  CubDebugExit(hipMemset(d_ht_p, 0, 2 * P_SLOT_LEN * sizeof(int)));
  CubDebugExit(hipMemset(d_ht_d, 0, 2 * D_SLOT_LEN * sizeof(int)));

  // 随机数生成器初始化，为了防止冲突需要设置多个。titan的并行度是72个sm * 32个warp/sm * 32个thread/warp
  hiprandState *curand_state;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&curand_state, MAX_CONCURRENT_THREADS * sizeof(hiprandState)));
  long clock_for_rand = clock();  //程序运行时钟数
  curandGenKernel<<<1, 1>>>(curand_state, clock_for_rand);    //main road

  cout << "TIMES: " << TIMES << "[";
  for (int i = 0; i < TIMES; i++) {
    if (i % 10 == 0) cout << i;
    cout << "=" << flush;
    {
      int items_per_tile = 128*4;
      int num_blocks = (C_LEN - 1) / items_per_tile + 1;
      build_hashtable_c<128,4><<<num_blocks, 128>>>(c_region, c_custkey, c_nation, d_ht_c);
      num_blocks = (S_LEN - 1) / items_per_tile + 1;
      build_hashtable_s<128,4><<<num_blocks, 128>>>(s_region, s_suppkey, d_ht_s);
      num_blocks = (P_LEN - 1) / items_per_tile + 1;
      build_hashtable_p<128,4><<<num_blocks, 128>>>(p_mfgr, p_partkey, d_ht_p);
      num_blocks = (D_LEN - 1) / items_per_tile + 1;
      build_hashtable_d<128,4><<<num_blocks, 128>>>(d_datekey, d_year, d_ht_d);
      num_blocks = (LO_LEN - 1) / THREADS_PER_BLOCK + 1;
      CubDebugExit(hipMemset(d_res, 0, res_array_size * sizeof(int)));
      createBSSampleAndProbe<THREADS_PER_BLOCK,1><<<num_blocks, THREADS_PER_BLOCK>>>(
          lo_orderdate, lo_custkey, lo_suppkey, lo_partkey, lo_revenue, lo_supplycost,
          d_ht_c, d_ht_s, d_ht_p, d_ht_d,
          d_res, curand_state);
      CubDebugExit(hipMemcpy(h_res, d_res, res_array_size * sizeof(int), hipMemcpyDeviceToHost));
      for (int i = 0; i < res_size; i++) {
        if (h_res[4*i] != 0) {
          int d_year = h_res[4*i];
          int c_nation = h_res[4*i + 1];
          long long sum = reinterpret_cast<long long*>(&h_res[4*i + 2])[0];
          pair<int, int> p(d_year, c_nation);
          (*bs_umap)[p].push_back(sum);
        }
      }
    }
  }
  cout << "]" << endl;
  for (auto& entry : *bs_umap) {
    vector<long long>& vec = entry.second;
    sort(vec.begin(), vec.end());
  }
  CLEANUP(d_ht_c);
  CLEANUP(d_ht_s);
  CLEANUP(d_ht_p);
  CLEANUP(d_ht_d);
  CLEANUP(curand_state);
}

void runQuery(
    int* lo_orderdate, int* lo_custkey, int* lo_suppkey, int* lo_partkey, int* lo_revenue, int* lo_supplycost,
    int* c_custkey, int* c_nation, int* c_region,
    int* s_suppkey, int* s_region,
    int* p_partkey, int* p_mfgr,
    int* d_datekey, int* d_year,
    unordered_map<pair<int, int>, long long, pair_hash>* umap) {

  // 存储单次查询结果
  int* d_res;
  int res_size = ((1998-1992+1) * 25);
  int res_array_size = res_size * 4;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_res, res_array_size * sizeof(int)));
  CubDebugExit(hipMemset(d_res, 0, res_array_size * sizeof(int)));
  int* h_res = new int[res_array_size];

  // 三张hash表
  int *d_ht_c, *d_ht_s, *d_ht_p, *d_ht_d;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_ht_c, 2 * C_SLOT_LEN * sizeof(int)));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_ht_s, 2 * S_SLOT_LEN * sizeof(int)));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_ht_p, 2 * P_SLOT_LEN * sizeof(int)));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_ht_d, 2 * D_SLOT_LEN * sizeof(int)));
  CubDebugExit(hipMemset(d_ht_c, 0, 2 * C_SLOT_LEN * sizeof(int)));
  CubDebugExit(hipMemset(d_ht_s, 0, 2 * S_SLOT_LEN * sizeof(int)));
  CubDebugExit(hipMemset(d_ht_p, 0, 2 * P_SLOT_LEN * sizeof(int)));
  CubDebugExit(hipMemset(d_ht_d, 0, 2 * D_SLOT_LEN * sizeof(int)));

  int items_per_tile = 128*4;
  int num_blocks = (C_LEN - 1) / items_per_tile + 1;
  build_hashtable_c<128,4><<<num_blocks, 128>>>(c_region, c_custkey, c_nation, d_ht_c);
  num_blocks = (S_LEN - 1) / items_per_tile + 1;
  build_hashtable_s<128,4><<<num_blocks, 128>>>(s_region, s_suppkey, d_ht_s);
  num_blocks = (P_LEN - 1) / items_per_tile + 1;
  build_hashtable_p<128,4><<<num_blocks, 128>>>(p_mfgr, p_partkey, d_ht_p);
  num_blocks = (D_LEN - 1) / items_per_tile + 1;
  build_hashtable_d<128,4><<<num_blocks, 128>>>(d_datekey, d_year, d_ht_d);
  num_blocks = (LO_LEN - 1) / items_per_tile + 1;
  probe<128,4><<<num_blocks, 128>>>(
      lo_orderdate, lo_custkey, lo_suppkey, lo_partkey, lo_revenue, lo_supplycost,
      d_ht_c, d_ht_s, d_ht_p, d_ht_d,
      d_res);
  CubDebugExit(hipMemcpy(h_res, d_res, res_array_size * sizeof(int), hipMemcpyDeviceToHost));
  for (int i = 0; i < res_size; i++) {
    if (h_res[4*i] != 0) {
      int d_year = h_res[4*i];
      int c_nation = h_res[4*i + 1];
      long long sum = reinterpret_cast<long long*>(&h_res[4*i + 2])[0];
      pair<int, int> p(d_year, c_nation);
      (*umap)[p] = sum;
    }
  }
  CLEANUP(d_ht_c);
  CLEANUP(d_ht_s);
  CLEANUP(d_ht_p);
  CLEANUP(d_ht_d);
}

void run(
    int* h_lo_orderdate, int* h_lo_custkey, int* h_lo_suppkey, int* h_lo_partkey, int* h_lo_revenue, int* h_lo_supplycost,
    int* h_c_custkey, int* h_c_nation, int* h_c_region,
    int* h_s_suppkey, int* h_s_region,
    int* h_p_partkey, int* h_p_mfgr,
    int* h_d_datekey, int* h_d_year,
    hipcub::CachingDeviceAllocator&  g_allocator) {
  // load column data to device memory
  int *d_lo_orderdate = loadToGPU<int>(h_lo_orderdate, LO_LEN, g_allocator);
  int *d_lo_custkey = loadToGPU<int>(h_lo_custkey, LO_LEN, g_allocator);
  int *d_lo_suppkey = loadToGPU<int>(h_lo_suppkey, LO_LEN, g_allocator);
  int *d_lo_partkey = loadToGPU<int>(h_lo_partkey, LO_LEN, g_allocator);
  int *d_lo_revenue = loadToGPU<int>(h_lo_revenue, LO_LEN, g_allocator);
  int *d_lo_supplycost = loadToGPU<int>(h_lo_supplycost, LO_LEN, g_allocator);
  int *d_c_custkey = loadToGPU<int>(h_c_custkey, C_LEN, g_allocator);
  int *d_c_nation = loadToGPU<int>(h_c_nation, C_LEN, g_allocator);
  int *d_c_region  = loadToGPU<int>(h_c_region, C_LEN, g_allocator);
  int *d_s_suppkey = loadToGPU<int>(h_s_suppkey, S_LEN, g_allocator);
  int *d_s_region = loadToGPU<int>(h_s_region, S_LEN, g_allocator);
  int *d_p_partkey = loadToGPU<int>(h_p_partkey, P_LEN, g_allocator);
  int *d_p_mfgr = loadToGPU<int>(h_p_mfgr, P_LEN, g_allocator);
  int *d_d_datekey = loadToGPU<int>(h_d_datekey, D_LEN, g_allocator);
  int *d_d_year = loadToGPU<int>(h_d_year, D_LEN, g_allocator);


	// 原始样本和bs样本的query值
  unordered_map<pair<int, int>, long long, pair_hash> umap;
  unordered_map<pair<int, int>, vector<long long>, pair_hash> bs_umap;

  // cuda注册start、stop事件，用于计时
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float time = 0.0f;
  hipEventRecord(start, 0);
  createBSSampleAndRunQuery(
      d_lo_orderdate, d_lo_custkey, d_lo_suppkey, d_lo_partkey, d_lo_revenue, d_lo_supplycost,
      d_c_custkey, d_c_nation, d_c_region,
      d_s_suppkey, d_s_region,
      d_p_partkey, d_p_mfgr,
      d_d_datekey, d_d_year,
      &bs_umap);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  cout << "Time Taken(resample + run query): " << time << "ms" << endl;

  // 计算原始样本的query值
  runQuery(
      d_lo_orderdate, d_lo_custkey, d_lo_suppkey, d_lo_partkey, d_lo_revenue, d_lo_supplycost,
      d_c_custkey, d_c_nation, d_c_region,
      d_s_suppkey, d_s_region,
      d_p_partkey, d_p_mfgr,
      d_d_datekey, d_d_year,
      &umap);

  for (auto& entry1 : umap) {
    auto p1 = entry1.first;
    long long sum = entry1.second;
    if(bs_umap.find(p1) == bs_umap.end()){
        cout << p1.first << "\t" << p1.second << "\t" << sum << "(0,0)" << endl;
      continue;
    }
    vector<long long> bs_sum = bs_umap[p1];
    int length = bs_sum.size();
    int idx1 = length * 0.01;
    int idx2 = length * 0.99;
    cout << p1.first << "\t" << p1.second << "\t";
    cout << sum << "(" << (double)(bs_sum[idx1]-sum)/sum << "," << (double)(bs_sum[idx2]-sum)/sum << ")" << endl;
  }
  cout << "Res Count: " << umap.size() << endl;

}
/**
 * Main
 */
int main(int argc, char** argv){
  // Initialize command line
  CommandLineArgs args(argc, argv);

  // Initialize device
  CubDebugExit(args.DeviceInit());

  // load column data to host memory
  int *h_lo_orderdate = loadColumn<int>("lo_orderdate", LO_LEN);
  int *h_lo_custkey = loadColumn<int>("lo_custkey", LO_LEN);
  int *h_lo_suppkey = loadColumn<int>("lo_suppkey", LO_LEN);
  int *h_lo_partkey = loadColumn<int>("lo_partkey", LO_LEN);
  int *h_lo_revenue = loadColumn<int>("lo_revenue", LO_LEN);
  int *h_lo_supplycost = loadColumn<int>("lo_supplycost", LO_LEN);
  int *h_c_custkey = loadColumn<int>("c_custkey", C_LEN);
  int *h_c_nation = loadColumn<int>("c_nation", C_LEN);
  int *h_c_region = loadColumn<int>("c_region", C_LEN);
  int *h_s_suppkey = loadColumn<int>("s_suppkey", S_LEN);
  int *h_s_region = loadColumn<int>("s_region", S_LEN);
  int *h_p_partkey = loadColumn<int>("p_partkey", P_LEN);
  int *h_p_mfgr = loadColumn<int>("p_mfgr", P_LEN);
  int *h_d_datekey = loadColumn<int>("d_datekey", D_LEN);
  int *h_d_year = loadColumn<int>("d_year", D_LEN);


  // 注册st、finish时间点，c++的计时工具
	chrono::high_resolution_clock::time_point st, finish;
  st = chrono::high_resolution_clock::now();
  run(
      h_lo_orderdate, h_lo_custkey, h_lo_suppkey, h_lo_partkey, h_lo_revenue, h_lo_supplycost,
      h_c_custkey, h_c_nation, h_c_region,
      h_s_suppkey, h_s_region,
      h_p_partkey, h_p_mfgr,
      h_d_datekey, h_d_year,
      g_allocator);
  finish = chrono::high_resolution_clock::now();
  chrono::duration<double> diff = finish - st;
  cout << "total time: " << diff.count() * 1000 << "ms" << endl;
  return 0;
}